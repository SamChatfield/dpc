#include "hip/hip_runtime.h"
//
// Sam Chatfield
// 1559986
//

#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define BLOCK_SIZE 1024

// A helper macro to simplify handling cuda error checking
#define CUDA_ERROR(err, msg) { \
if (err != hipSuccess) {\
    printf( "%s: %s in %s at line %d\n", msg, hipGetErrorString( err ), __FILE__, __LINE__);\
    exit( EXIT_FAILURE );\
}\
}

// Host sequential version of block reduce using a single thread
__host__ void host_blk_reduce(int numElements, int blockSize, float *A)
{
	int segSize = blockSize * 2;
	int numSeg = ceil((float) numElements / (float) segSize);
	// Loop over each segment
	for (int segNum = 0; segNum < numSeg; segNum++)
	{
		int segStartIdx = segNum * segSize;
		// Loop over each element in this segment starting at 1 (0 is the accumulator)
		for (int i = 1; i < segSize; i++)
		{
			int idx = segStartIdx + i;
			if (idx > numElements - 1)
			{
				break;
			}
			A[segStartIdx] += A[idx];
		}
	}
}

// Kernel sequential version of block reduce using a single thread
__global__ void single_thread_blk_reduce(int numElements, int blockSize, float *A)
{
	int segSize = blockSize * 2;
	int numSeg = ceil((float) numElements / (float) segSize);
	// Loop over each segment
	for (int segNum = 0; segNum < numSeg; segNum++)
	{
		int segStartIdx = segNum * segSize;
		// Loop over each element in this segment starting at 1 (0 is the accumulator)
		for (int i = 1; i < segSize; i++)
		{
			int idx = segStartIdx + i;
			if (idx > numElements - 1)
			{
				break;
			}
			A[segStartIdx] += A[idx];
		}
	}
}

// Kernel parallel version of block reduce using global memory
__global__ void global_blk_reduce(int numElements, float *A)
{
	// The size of a data segment is 2 x blockSize
	int segSize = blockDim.x * 2;

	// Work out the index of the vector that this thread is working on
	int i = blockIdx.x * segSize + threadIdx.x;

	if (i < numElements)
	{
		for (uint stride = blockDim.x; stride > 0; stride /= 2)
		{
			__syncthreads();
//			if (blockIdx.x == 0 && threadIdx.x == 0)
//			{
//				printf("stride: %d\n", stride);
//				printf("A[%d] (%0.1f) += A[%d + %d] (%0.1f)\n", i, A[i], i, stride, A[i + stride]);
//			}
			if (threadIdx.x < stride && i + stride < numElements)
			{
				A[i] += A[i + stride];
			}
//			if (blockIdx.x == 0 && threadIdx.x == 0)
//			{
//				printf("A[%d] = %0.1f\n", i, A[i]);
//			}
		}
	}
}

// Kernel parallel version of block reduce using shared memory
__global__ void shared_blk_reduce(int numElements, float *A)
{
	// The size of a data segment is 2 x blockSize
	int segSize = blockDim.x * 2;

	// Work out the index of the vector that this thread is working on
	int i = blockIdx.x * segSize + threadIdx.x;

	// Allocate space for the current segment in device shared memory
	__device__ __shared__ float segment[BLOCK_SIZE * 2];

	// Copy the element to the shared vector
	if (i < numElements)
	{
		segment[threadIdx.x] = A[i];
		segment[threadIdx.x + blockDim.x] = A[i + blockDim.x];

		for (uint stride = blockDim.x; stride > 0; stride /= 2)
		{
			__syncthreads();
//			if (blockIdx.x == 0 && threadIdx.x == 0)
//			{
//				printf("stride: %d\n", stride);
//				printf("segment[%d] (%0.1f) += segment[%d + %d] (%0.1f)\n", threadIdx.x, segment[threadIdx.x], threadIdx.x, stride, segment[threadIdx.x + stride]);
//			}
			if (threadIdx.x < stride && i + stride < numElements)
			{
				segment[threadIdx.x] += segment[threadIdx.x + stride];
			}
//			if (blockIdx.x == 0 && threadIdx.x == 0)
//			{
//				printf("segment[%d] = %0.1f\n", threadIdx.x, segment[threadIdx.x]);
//			}
		}
	}

	// Copy only the result back into the global vector
	if (threadIdx.x == 0)
	{
		A[i] = segment[0];
	}
}

// Host sequential version of full reduce using a single thread
__host__ void host_full_reduce(int numElements, float *A)
{
	for (int i = 1; i < numElements; i++)
	{
		A[0] += A[i];
	}
	printf("Sum for host_full_reduce = %0.1f\n", A[0]);
}

__global__ void full_reduce_compress(int numSegments, int segSize, float *A)
{
	int seg = blockIdx.x * blockDim.x + threadIdx.x;

	if (seg < numSegments)
	{
		A[seg] = A[seg * segSize];
	}
}

// Kernel parallel version of full reduce using global memory
__host__ void global_full_reduce(int numElements, int numSegments, int blockSize, float *A)
{
	int remainingElements = numElements;
	int remainingSegments = numSegments;

	while (remainingElements > 1)
	{
		global_blk_reduce<<<remainingSegments, blockSize>>>(remainingElements, A);

		remainingElements = remainingSegments;
		remainingSegments = 1 + ((remainingElements - 1) / (blockSize * 2));

		full_reduce_compress<<<remainingSegments * 2, blockSize>>>(remainingElements, blockSize * 2, A);
	}
}

// Kernel parallel version of full reduce using shared memory
__host__ void shared_full_reduce(int numElements, int numSegments, int blockSize, float *A)
{
	int remainingElements = numElements;
	int remainingSegments = numSegments;

	while (remainingElements > 1)
	{
		shared_blk_reduce<<<remainingSegments, blockSize>>>(remainingElements, A);

		remainingElements = remainingSegments;
		remainingSegments = 1 + ((remainingElements - 1) / (blockSize * 2));

		full_reduce_compress<<<remainingSegments * 2, blockSize>>>(remainingElements, blockSize * 2, A);
	}
}

// Compare two vectors for equality as expected by block reduce
void compare_results_blk(float *A, float *B, int numElements, int blockSize)
{
	int segSize = blockSize * 2;
	for (int i = 0; i < numElements; i += segSize)
	{
		if (A[i] != B[i])
		{
			printf("TEST FAILED at element %d where A=%0.1f and B=%0.1f\n", i, A[i], B[i]);
			exit(EXIT_FAILURE);
		}
	}
	printf("Test passed\n");
}

// Check that the result in the first element of the vector is the number of elements in the vector
void compare_results_full(int numElements, float *A)
{
	if (A[0] != numElements)
	{
		printf("TEST FAILED at element 0 where A=%0.1f and numElements=%d\n", A[0], numElements);
		exit(EXIT_FAILURE);
	}
	printf("Test passed\n");
}

int main()
{
	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	// Create host timer
	StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);
	double h_msecs;

	// Create Device timer event objects
	hipEvent_t start, stop;
	float d_msecs;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Use this to disable the single_thread_blk_reduce kernel which hugely slows down the computation for large vectors
	int doSingleThread = true;

	int blockSize = 1024;
	int numElements = 1000000;
//	int numElements = 10000000;
	int numBlocks = 1 + ((numElements - 1) / blockSize);
	int numSegments = 1 + ((numElements - 1) / (blockSize * 2));
	printf("numBlocks=%d, numSegments=%d\n", numBlocks, numSegments);
	size_t size = numElements * sizeof(float);
	printf("[Sum Reduce of %d elements with %d blocks of size %d]\n", numElements, numBlocks, blockSize);

	// Allocate host vector A
	float *h_A = (float*) malloc(size);

	// Verify that allocations succeeded
	if (h_A == NULL)
	{
		fprintf(stderr, "Failed to allocate host vector h_A\n");
		exit(EXIT_FAILURE);
	}

	// Initialise the host vector
	for (int i = 0; i < numElements; i++)
	{
		h_A[i] = 1.0f;
	}

	//
	// HOST_BLK_REDUCE
	//

	// Allocate memory on host for host_blk_reduce, and copy values from h_A
	float *h_A_hbr = (float*) malloc(size);
	memcpy(h_A_hbr, h_A, size);

	sdkStartTimer(&timer);
	host_blk_reduce(numElements, blockSize, h_A_hbr);
	sdkStopTimer(&timer);

	h_msecs = sdkGetTimerValue(&timer);
	printf("[HOST_BLK_REDUCE] Executed sum reduce of %d elements on the host in = %.5fmSecs\n", numElements, h_msecs);

	//
	// SINGLE_THREAD_BLK_REDUCE
	//

	if (doSingleThread == true)
	{
		// Allocate memory on device for single_thread_blk_reduce, and copy values from h_A
		float *d_A_stbr = NULL;
		err = hipMalloc((void**) &d_A_stbr, size);
		CUDA_ERROR(err, "Failed to allocate device vector d_A_stbr");
		err = hipMemcpy(d_A_stbr, h_A, size, hipMemcpyHostToDevice);
		CUDA_ERROR(err, "Failed to copy vector h_A to d_A_stbr");

		hipEventRecord(start, 0);
		single_thread_blk_reduce<<<1, 1>>>(numElements, blockSize, d_A_stbr);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);

		// Wait for device to finish
		hipDeviceSynchronize();

		err = hipGetLastError();
		CUDA_ERROR(err, "Failed to launch single_thread_blk_reduce kernel");

		err = hipEventElapsedTime(&d_msecs, start, stop);
		CUDA_ERROR(err, "Failed to get elapsed time");
		printf("[SINGLE_THREAD_BLK_REDUCE] Executed sum reduce of %d elements on device in a single thread in = %.5fmSecs\n", numElements, d_msecs);

		// Verify result against result of host_blk_reduce
		float *h_A_stbr = (float*) malloc(size);
		err = hipMemcpy(h_A_stbr, d_A_stbr, size, hipMemcpyDeviceToHost);
		CUDA_ERROR(err, "Failed to copy vector d_A_stbr to h_A_stbr");
		compare_results_blk(h_A_hbr, h_A_stbr, numElements, blockSize);
		free(h_A_stbr);

		err = hipFree(d_A_stbr);
		CUDA_ERROR(err, "Failed to free device vector d_A_stbr");
	}

	//
	// GLOBAL_BLK_REDUCE
	//

	// Allocate memory on device for global_blk_reduce, and copy values from h_A
	float *d_A_gbr = NULL;
	err = hipMalloc((void**) &d_A_gbr, size);
	CUDA_ERROR(err, "Failed to allocate device vector d_A_gbr");
	err = hipMemcpy(d_A_gbr, h_A, size, hipMemcpyHostToDevice);
	CUDA_ERROR(err, "Failed to copy vector h_A to d_A_gbr");

	hipEventRecord(start, 0);
	global_blk_reduce<<<numSegments, blockSize>>>(numElements, d_A_gbr);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	// Wait for device to finish
	hipDeviceSynchronize();

	err = hipGetLastError();
	CUDA_ERROR(err, "Failed to launch global_blk_reduce kernel");

	err = hipEventElapsedTime(&d_msecs, start, stop);
	CUDA_ERROR(err, "Failed to get elapsed time");
	printf("[GLOBAL_BLK_REDUCE] Executed sum reduce of %d elements on device in %d blocks of %d threads in = %.5fmSecs\n",
			numElements, numSegments, blockSize, d_msecs);

	// Verify result against result of host_blk_reduce
	float *h_A_gbr = (float*) malloc(size);
	err = hipMemcpy(h_A_gbr, d_A_gbr, size, hipMemcpyDeviceToHost);
	CUDA_ERROR(err, "Failed to copy vector d_A_gbr to h_A_gbr");
	compare_results_blk(h_A_hbr, h_A_gbr, numElements, blockSize);
	free(h_A_gbr);

	err = hipFree(d_A_gbr);
	CUDA_ERROR(err, "Failed to free device vector d_A_gbr");

	//
	// SHARED_BLK_REDUCE
	//

	// Allocate memory on device for shared_blk_reduce, and copy values from h_A
	float *d_A_sbr = NULL;
	err = hipMalloc((void**) &d_A_sbr, size);
	CUDA_ERROR(err, "Failed to allocate device vector d_A_sbr");
	err = hipMemcpy(d_A_sbr, h_A, size, hipMemcpyHostToDevice);
	CUDA_ERROR(err, "Failed to copy vector h_A to d_A_sbr");

	hipEventRecord(start, 0);
	shared_blk_reduce<<<numSegments, blockSize>>>(numElements, d_A_sbr);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	// Wait for device to finish
	hipDeviceSynchronize();

	err = hipGetLastError();
	CUDA_ERROR(err, "Failed to launch shared_blk_reduce kernel");

	err = hipEventElapsedTime(&d_msecs, start, stop);
	CUDA_ERROR(err, "Failed to get elapsed time");
	printf("[SHARED_BLK_REDUCE] Executed sum reduce of %d elements on device in %d blocks of %d threads in = %.5fmSecs\n",
			numElements, numSegments, blockSize, d_msecs);

	// Verify result against result of host_blk_reduce
	float *h_A_sbr = (float*) malloc(size);
	err = hipMemcpy(h_A_sbr, d_A_sbr, size, hipMemcpyDeviceToHost);
	CUDA_ERROR(err, "Failed to copy vector d_A_sbr to h_A_sbr");
	compare_results_blk(h_A_hbr, h_A_sbr, numElements, blockSize);
	free(h_A_sbr);

	err = hipFree(d_A_sbr);
	CUDA_ERROR(err, "Failed to free device vector d_A_sbr");

	//
	// BLK_REDUCE TEARDOWN
	//

	free(h_A_hbr);

	//
	// HOST_FULL_REDUCE
	//

	// Allocate memory on host for host_blk_reduce, and copy values from h_A
	float *h_A_hfr = (float*) malloc(size);
	if (h_A_hfr == NULL)
	{
		fprintf(stderr, "Failed to allocate host vector h_A_hfr\n");
		exit(EXIT_FAILURE);
	}
	memcpy(h_A_hfr, h_A, size);

	sdkStartTimer(&timer);
	host_full_reduce(numElements, h_A_hfr);
	sdkStopTimer(&timer);

	h_msecs = sdkGetTimerValue(&timer);
	printf("[HOST_FULL_REDUCE] Executed sum reduce of %d elements on the host in = %.5fmSecs\n", numElements, h_msecs);

	free(h_A_hfr);

	//
	// GLOBAL_FULL_REDUCE
	//

	// Allocate memory on device for global_blk_reduce, and copy values from h_A
	float *d_A_gfr = NULL;
	err = hipMalloc((void**) &d_A_gfr, size);
	CUDA_ERROR(err, "Failed to allocate device vector d_A_gfr");
	err = hipMemcpy(d_A_gfr, h_A, size, hipMemcpyHostToDevice);
	CUDA_ERROR(err, "Failed to copy vector h_A to d_A_gfr");

	hipEventRecord(start, 0);
	global_full_reduce(numElements, numSegments, blockSize, d_A_gfr);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	err = hipEventElapsedTime(&d_msecs, start, stop);
	CUDA_ERROR(err, "Failed to get elapsed time");
	printf("[GLOBAL_FULL_REDUCE] Executed sum reduce of %d elements on device in = %.5fmSecs\n",
			numElements, numSegments, blockSize, d_msecs);

	// Verify result against numElements
	float *h_A_gfr = (float*) malloc(size);
	err = hipMemcpy(h_A_gfr, d_A_gfr, size, hipMemcpyDeviceToHost);
	CUDA_ERROR(err, "Failed to copy vector d_A_gfr to h_A_gfr");
	compare_results_full(numElements, h_A_gfr);
	free(h_A_gfr);

	err = hipFree(d_A_gfr);
	CUDA_ERROR(err, "Failed to free device vector d_A_gfr");

	//
	// SHARED_FULL_REDUCE
	//

	// Allocate memory on device for global_blk_reduce, and copy values from h_A
	float *d_A_sfr = NULL;
	err = hipMalloc((void**) &d_A_sfr, size);
	CUDA_ERROR(err, "Failed to allocate device vector d_A_sfr");
	err = hipMemcpy(d_A_sfr, h_A, size, hipMemcpyHostToDevice);
	CUDA_ERROR(err, "Failed to copy vector h_A to d_A_sfr");

	hipEventRecord(start, 0);
	shared_full_reduce(numElements, numSegments, blockSize, d_A_sfr);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	err = hipEventElapsedTime(&d_msecs, start, stop);
	CUDA_ERROR(err, "Failed to get elapsed time");
	printf("[SHARED_FULL_REDUCE] Executed sum reduce of %d elements on device in = %.5fmSecs\n",
			numElements, numSegments, blockSize, d_msecs);

	// Verify result against numElements
	float *h_A_sfr = (float*) malloc(size);
	err = hipMemcpy(h_A_sfr, d_A_sfr, size, hipMemcpyDeviceToHost);
	CUDA_ERROR(err, "Failed to copy vector d_A_sfr to h_A_sfr");
	compare_results_full(numElements, h_A_sfr);
	free(h_A_sfr);

	err = hipFree(d_A_sfr);
	CUDA_ERROR(err, "Failed to free device vector d_A_sfr");

	//
	// TEARDOWN
	//

	// Free host memory
	free(h_A);

	// Clean up the Host timer
	sdkDeleteTimer(&timer);

	// Clean up the Device timer event objects
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Reset the device and exit
	err = hipDeviceReset();
	CUDA_ERROR(err, "Failed to reset the device");

	printf("Done\n");
	return EXIT_SUCCESS;
}
