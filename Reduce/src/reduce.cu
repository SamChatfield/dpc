#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define BLOCK_SIZE 1024

// A helper macro to simplify handling cuda error checking
#define CUDA_ERROR( err, msg ) { \
if (err != hipSuccess) {\
    printf( "%s: %s in %s at line %d\n", msg, hipGetErrorString( err ), __FILE__, __LINE__);\
    exit( EXIT_FAILURE );\
}\
}

// Host sequential version of block reduce using a single thread
__host__ void host_blk_reduce(int numElements, int blockSize, float *A)
{
	int segSize = blockSize * 2;
	int numSeg = ceil((float) numElements / (float) segSize);
	printf("%d segments of size %d for %d elements\n", numSeg, segSize, numElements);
	// Loop over each segment
	for (int segNum = 0; segNum < numSeg; segNum++)
	{
		int segStartIdx = segNum * segSize;
//		printf("segStartIdx = %d\n", segStartIdx);
		// Loop over each element in this segment starting at 1 (0 is the accumulator)
		for (int i = 1; i < segSize - 1; i++)
		{
			int idx = segStartIdx + i;
			if (idx > numElements - 1)
			{
//				printf("Segment finished early at i=%d\n", i);
				break;
			}
//			printf("Add i=%d: %f to acc i=%d: %f\n", idx, A[idx], segStartIdx, A[segStartIdx]);
			A[segStartIdx] += A[idx];
		}
		printf("Sum for segment %d = %f\n", segNum, A[segStartIdx]);
	}
}

// Kernel sequential version of block reduce using a single thread
__global__ void single_thread_blk_reduce(int numElements, int blockSize, float *A)
{
	int segSize = blockSize * 2;
	int numSeg = ceil((float) numElements / (float) segSize);
	printf("%d segments of size %d for %d elements\n", numSeg, segSize, numElements);
	// Loop over each segment
	for (int segNum = 0; segNum < numSeg; segNum++)
	{
		int segStartIdx = segNum * segSize;
//		printf("segStartIdx = %d\n", segStartIdx);
		// Loop over each element in this segment starting at 1 (0 is the accumulator)
		for (int i = 1; i < segSize - 1; i++)
		{
			int idx = segStartIdx + i;
			if (idx > numElements - 1)
			{
//				printf("Segment finished early at i=%d\n", i);
				break;
			}
//			printf("Add i=%d: %f to acc i=%d: %f\n", idx, A[idx], segStartIdx, A[segStartIdx]);
			A[segStartIdx] += A[idx];
		}
//		printf("Sum for segment %d = %f\n", segNum, A[segStartIdx]);
	}
}

// Kernel parallel version of block reduce using global memory
__global__ void global_blk_reduce(int numElements, float *A)
{
	// The size of a data segment is 2 x blockSize
	int segSize = blockDim.x * 2;

	// Work out the index of the vector that this thread is working on
	int i = blockIdx.x * segSize + threadIdx.x;

	for (uint stride = blockDim.x; stride > 0; stride /= 2)
	{
		__syncthreads();
		// TODO: remove
//		if (blockIdx.x == 24 && threadIdx.x == 0)
//		{
//			printf("stride: %d\n", stride);
//			printf("A[%d] (%f) += A[%d + %d] (%f)\n", i, A[i], i, stride, A[i + stride]);
//		}
		if (threadIdx.x < stride)
		{
			A[i] += A[i + stride];
		}
//		if (blockIdx.x == 24 && threadIdx.x == 0)
//		{
//			printf("A[%d] = %f\n", i, A[i]);
//		}
	}
}

// Kernel parallel version of block reduce using shared memory
__global__ void shared_blk_reduce(int numElements, float *A)
{
	// The size of a data segment is 2 x blockSize
	int segSize = blockDim.x * 2;

	// Work out the index of the vector that this thread is working on
	int i = blockIdx.x * segSize + threadIdx.x;

	// Allocate space for the current segment in device shared memory
	__device__ __shared__ float segment[BLOCK_SIZE * 2];

	// Copy the element to the shared vector
	if (i < numElements)
	{
		segment[threadIdx.x] = A[i];
		segment[threadIdx.x + blockDim.x] = A[i + blockDim.x];
	}

	for (uint stride = blockDim.x; stride > 0; stride /= 2)
	{
		__syncthreads();
		// TODO: remove
//		if (blockIdx.x == 0 && threadIdx.x == 0)
//		{
//			printf("stride: %d\n", stride);
//			printf("segment[%d] (%f) += segment[%d + %d] (%f)\n", threadIdx.x, segment[threadIdx.x], threadIdx.x, stride, segment[threadIdx.x + stride]);
//		}
		if (threadIdx.x < stride)
		{
			segment[threadIdx.x] += segment[threadIdx.x + stride];
		}
//		if (blockIdx.x == 0 && threadIdx.x == 0)
//		{
//			printf("segment[%d] = %f\n", threadIdx.x, segment[threadIdx.x]);
//		}
	}

	// Copy only the result back into the global vector
	if (threadIdx.x == 0)
	{
		A[i] = segment[0];
	}
}

// Host sequential version of full reduce using a single thread
__host__ void host_full_reduce()
{

}

// Kernel parallel version of full reduce using global memory
__global__ void global_full_reduce()
{

}

// Kernel parallel version of full reduce using shared memory
__global__ void shared_full_reduce()
{

}

int main()
{
	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	// Create host timer
	StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);
	double h_msecs;

	// Create Device timer event objects
	hipEvent_t start, stop;
	float d_msecs;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int blockSize = 1024;
//	int numElements = 51200;
	int numElements = 50000;
	int numBlocks = 1 + ((numElements - 1) / blockSize);
	int numSegments = 1 + ((numElements - 1) / (blockSize * 2));
	printf("numBlocks=%d, numSegments=%d\n", numBlocks, numSegments);
	size_t size = numElements * sizeof(float);
	printf("[Sum Reduce of %d elements with %d blocks of size %d]\n", numElements, numBlocks, blockSize);

	// Allocate host vector A
	float *h_A = (float*) malloc(size);

	// Verify that allocations succeeded
	if (h_A == NULL)
	{
		fprintf(stderr, "Failed to allocate host vector h_A\n");
		exit(EXIT_FAILURE);
	}

	// Initialise the host vector
	for (int i = 0; i < numElements; i++)
	{
		h_A[i] = 1.0f;
	}

	// Allocate the device vector A
	float *d_A = NULL;
	err = hipMalloc((void**) &d_A, size);
	CUDA_ERROR(err, "Failed to allocate device vector d_A");

	// Initialise the device vector by copying from the host vector
	printf("Copy input data from the host memory to the CUDA device\n");
	err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	CUDA_ERROR(err, "Failed to copy vector A from host to device");

	//
	// host_blk_reduce
	//

	sdkStartTimer(&timer);
	host_blk_reduce(numElements, blockSize, h_A);
	sdkStopTimer(&timer);

	h_msecs = sdkGetTimerValue(&timer);
	printf("[host_blk_reduce] Executed Sum Reduce of %d elements on the Host in = %.5fmSecs\n", numElements, h_msecs);

	//
	// single_thread_blk_reduce
	//

	// Allocate memory on device for single_thread_blk_reduce, and copy values from d_A
	float *d_A_stbr = NULL;
	err = hipMalloc((void**) &d_A_stbr, size);
	CUDA_ERROR(err, "Failed to allocate device vector d_A_stbr");
	err = hipMemcpy(d_A_stbr, d_A, size, hipMemcpyDeviceToDevice);
	CUDA_ERROR(err, "Failed to copy vector d_A to d_A_stbr");

	hipEventRecord(start, 0);
	single_thread_blk_reduce<<<1, 1>>>(numElements, blockSize, d_A_stbr);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	// Wait for device to finish
	hipDeviceSynchronize();

	err = hipGetLastError();
	CUDA_ERROR(err, "Failed to launch single_thread_blk_reduce kernel");

	err = hipEventElapsedTime(&d_msecs, start, stop);
	CUDA_ERROR(err, "Failed to get elapsed time");
	printf("[single_thread_blk_reduce] Executed Sum Reduce of %d elements on the Device in a SINGLE THREAD in = %.5fmSecs\n", numElements, d_msecs);

	err = hipFree(d_A_stbr);
	CUDA_ERROR(err, "Failed to free device vector d_A_stbr");

	//
	// global_blk_reduce
	//

	// Allocate memory on device for global_blk_reduce, and copy values from d_A
	float *d_A_gbr = NULL;
	err = hipMalloc((void**) &d_A_gbr, size);
	CUDA_ERROR(err, "Failed to allocate device vector d_A_gbr");
	err = hipMemcpy(d_A_gbr, d_A, size, hipMemcpyDeviceToDevice);
	CUDA_ERROR(err, "Failed to copy vector d_A to d_A_gbr");

	hipEventRecord(start, 0);
	global_blk_reduce<<<numSegments, blockSize>>>(numElements, d_A_gbr);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	// Wait for device to finish
	hipDeviceSynchronize();

	err = hipGetLastError();
	CUDA_ERROR(err, "Failed to launch global_blk_reduce kernel");

	err = hipEventElapsedTime(&d_msecs, start, stop);
	CUDA_ERROR(err, "Failed to get elapsed time");
	printf("[global_blk_reduce] Executed Sum Reduce of %d elements on the Device in %d blocks of %d threads in = %.5fmSecs\n",
			numElements, numSegments, blockSize, d_msecs);

	err = hipFree(d_A_gbr);
	CUDA_ERROR(err, "Failed to free device vector d_A_gbr");

	//
	// shared_blk_reduce
	//

	// Allocate memory on device for global_blk_reduce, and copy values from d_A
	float *d_A_sbr = NULL;
	err = hipMalloc((void**) &d_A_sbr, size);
	CUDA_ERROR(err, "Failed to allocate device vector d_A_sbr");
	err = hipMemcpy(d_A_sbr, d_A, size, hipMemcpyDeviceToDevice);
	CUDA_ERROR(err, "Failed to copy vector d_A to d_A_sbr");

	hipEventRecord(start, 0);
	shared_blk_reduce<<<numSegments, blockSize>>>(numElements, d_A_sbr);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	// Wait for device to finish
	hipDeviceSynchronize();

	err = hipGetLastError();
	CUDA_ERROR(err, "Failed to launch shared_blk_reduce kernel");

	err = hipEventElapsedTime(&d_msecs, start, stop);
	CUDA_ERROR(err, "Failed to get elapsed time");
	printf("[shared_blk_reduce] Executed Sum Reduce of %d elements on the Device in %d blocks of %d threads in = %.5fmSecs\n",
			numElements, numSegments, blockSize, d_msecs);

	err = hipFree(d_A_sbr);
	CUDA_ERROR(err, "Failed to free device vector d_A_sbr");

	//
	// Teardown
	//

	// Free device global memory
	err = hipFree(d_A);
	CUDA_ERROR(err, "Failed to free device vector d_A");

	// Free host memory
	free(h_A);

	// Clean up the Host timer
	sdkDeleteTimer(&timer);

	// Clean up the Device timer event objects
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Reset the device and exit
	err = hipDeviceReset();
	CUDA_ERROR(err, "Failed to reset the device");

	printf("Done\n");
	return 0;
}
