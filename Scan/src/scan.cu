#include "hip/hip_runtime.h"
//
// Sam Chatfield
// 1559986
//
// Intel Core i5-6500
// GeForce GTX 960
//

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define BLOCK_SIZE 1024

// A helper macro to simplify handling cuda error checking
#define CUDA_ERROR(err, msg) { \
if (err != hipSuccess) {\
    printf( "%s: %s in %s at line %d\n", msg, hipGetErrorString( err ), __FILE__, __LINE__);\
    exit( EXIT_FAILURE );\
}\
}

void h_block_scan(int n, int numBlocks, int blockSize, int *in, int *out)
{
	for (int b = 0; b < numBlocks; b++)
	{
		out[b * blockSize] = 0;
		for (int i = 1; i < blockSize; i++)
		{

			int elem = b * blockSize + i;
			if (elem < n)
			{
				out[elem] = in[elem-1] + out[elem-1];
			}
		}
	}
}

__global__ void d_block_scan(int n, int *in, int *out, int *sums)
{
	__shared__ int temp[BLOCK_SIZE * 2];

	int segSize = blockDim.x * 2;
	int thid = threadIdx.x;
	int segStartIdx = segSize * blockIdx.x;
	int offset = 1;

	// Load input into shared memory
	if (segStartIdx + 2*thid < n)
	{
		temp[2*thid] = in[segStartIdx + 2*thid];
		temp[2*thid+1] = in[segStartIdx + 2*thid+1];
	}
	else{
		temp[2*thid] = 0;
		temp[2*thid+1] = 0;
	}

	// Build sum in place up the tree
	for (int d = segSize >> 1; d > 0; d >>= 1)
	{
		__syncthreads();

		if (thid < d)
		{
			int ai = offset * (2*thid+1) - 1;
			int bi = offset * (2*thid+2) - 1;

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	// Move total sum to sums array
	sums[blockIdx.x] = temp[segSize-1];

	// Zero the last element
	if (thid == 0) { temp[segSize-1] = 0; };

	// Traverse down tree and build scan
	for (int d = 1; d < segSize; d *= 2)
	{
		offset >>= 1;
		__syncthreads();

		if (thid < d)
		{
			int ai = offset * (2*thid+1) - 1;
			int bi = offset * (2*thid+2) - 1;

			float t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}

	__syncthreads();

	// Write results to global memory
	if (segStartIdx + 2*thid < n)
	{
		out[segStartIdx + 2*thid] = temp[2*thid];
		out[segStartIdx + 2*thid+1] = temp[2*thid+1];
	}
}

void h_full_scan(int numElements, int *in, int *out)
{
	out[0] = 0;
	for (int i = 1; i < numElements; i++)
	{
		out[i] = in[i-1] + out[i-1];
	}
}

__global__ void d_uniform_add(int n, int *out, int *incr)
{
	// TODO: Try storing one value of incr in shared mem

	int segSize = blockDim.x * 2;
	int thid = threadIdx.x;
	int segStartIdx = segSize * blockIdx.x;

	if (segStartIdx + 2*thid < n)
	{
		out[segStartIdx + 2*thid] = incr[blockIdx.x];
		out[segStartIdx + 2*thid+1] = incr[blockIdx.x];
	}
}

void one_level_scan(int n, int numSegments, int segSize, int *in, int *out)
{
	hipError_t err = hipSuccess;

	size_t sumsSize = numSegments * sizeof(int);
	int *sums = NULL;
	err = hipMalloc((void**) &sums, sumsSize);
	CUDA_ERROR(err, "Failed to allocate device vector sums");

	d_block_scan<<<numSegments, segSize/2>>>(n, in, out, sums);
	hipDeviceSynchronize();
	err = hipGetLastError();
	CUDA_ERROR(err, "Failed to launch d_block_scan kernel");

	err = hipFree(sums);
	CUDA_ERROR(err, "Failed to free device vector sums");
}

void two_level_scan(int n, int numSegments, int segSize, int *in, int *out)
{
	hipError_t err = hipSuccess;

	size_t sumsSize = numSegments * sizeof(int);
	int *sums = NULL;
	err = hipMalloc((void**) &sums, sumsSize);
	CUDA_ERROR(err, "Failed to allocate device vector sums");

	d_block_scan<<<numSegments, segSize/2>>>(n, in, out, sums);
	hipDeviceSynchronize();
	err = hipGetLastError();
	CUDA_ERROR(err, "Failed to launch d_block_scan kernel");

	// Debug sums output
	int *h_SUMS_dbs = (int*) malloc(sumsSize);
	err = hipMemcpy(h_SUMS_dbs, sums, sumsSize, hipMemcpyDeviceToHost);
	CUDA_ERROR(err, "Failed to copy vector sums to h_SUMS_dbs");
	for (int i = 0; i < numSegments; i++)
	{
		printf("sums[%d] = %d\n", i, h_SUMS_dbs[i]);
	}
	free(h_SUMS_dbs);

	int *incr = NULL;
	err = hipMalloc((void**) &incr, sumsSize);
	CUDA_ERROR(err, "Failed to allocate device vector incr");

	int *sums2 = NULL;
	err = hipMalloc((void**) &sums2, sizeof(int));
	CUDA_ERROR(err, "Failed to allocate device vector sums");

	d_block_scan<<<1, numSegments/2>>>(n, sums, incr, sums2);
	hipDeviceSynchronize();
	err = hipGetLastError();
	CUDA_ERROR(err, "Failed to launch d_block_scan kernel");

	// Debug incr output
	int *h_INCR_dbs = (int*) malloc(sumsSize);
	err = hipMemcpy(h_INCR_dbs, incr, sumsSize, hipMemcpyDeviceToHost);
	CUDA_ERROR(err, "Failed to copy vector sums to h_INCR_dbs");
	for (int i = 0; i < numSegments; i++)
	{
		printf("incr[%d] = %d\n", i, h_INCR_dbs[i]);
	}
	free(h_INCR_dbs);

	d_uniform_add<<<numSegments, segSize/2>>>(n, out, incr);
	hipDeviceSynchronize();
	err = hipGetLastError();
	CUDA_ERROR(err, "Failed to launch d_uniform_add kernel");

	err = hipFree(sums);
	CUDA_ERROR(err, "Failed to free device vector sums");
	err = hipFree(sums2);
	CUDA_ERROR(err, "Failed to free device vector sums2");
	err = hipFree(incr);
	CUDA_ERROR(err, "Failed to free device vector incr");
}

void three_level_scan(int n, int *in, int *out)
{

}

void d_full_scan(int n, int numSegments, int segSize, int *in, int *out)
{
	if (n <= segSize)
	{
		// 1 Level Scan
		printf("[D_FULL_SCAN] Full scan %d elements with segment size %d => 1 level scan\n", n, segSize);
		one_level_scan(n, numSegments, segSize, in, out);
	}
	else if (n > segSize && n <= segSize * segSize)
	{
		// 2 Level Scan
		printf("[D_FULL_SCAN] Full scan %d elements with segment size %d => 2 level scan\n", n, segSize);
		two_level_scan(n, numSegments, segSize, in, out);
	}
	else if (n > segSize * segSize)
	{
		// 3 Level Scan
		printf("[D_FULL_SCAN] Full scan %d elements with segment size %d => 3 level scan\n", n, segSize);
	}
	else {
		printf("Invalid number of elements %d", n);
		exit(EXIT_FAILURE);
	}
}

bool correct_results_block(int length, int blockSize, int *result, int *expected)
{
	return false;
}

bool correct_results_full(int length, int *result, int *expected)
{
	for (int i = 0; i < length; i++)
	{
		if (result[i] != expected[i])
		{
			printf("TEST FAILED at element %d: %d received, %d expected\n", i, result[i], expected[i]);
			return false;
		}
	}
	return true;
}

int main()
{
	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	// Create host timer
	StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);
	double h_msecs;

	// Create Device timer event objects
	hipEvent_t start, stop;
	float d_msecs;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int blockSize = 1024;
	int segSize = blockSize * 2;
//	int numElements = 1024;
	int numElements = 1048576;
	size_t size = numElements * sizeof(int);
	int numBlocks = 1 + ((numElements - 1) / blockSize);
	int numSegments = 1 + ((numElements - 1) / (segSize));

	// Create and initialise host input vector
	int *h_A = (int*) malloc(size);
	printf("Sum Scan of %d elements\n", numElements);

	for (int i = 0; i < numElements; i++)
	{
		h_A[i] = rand() % 10;
	}

	// Copy host input vector to device
	int *d_A = NULL;
	err = hipMalloc((void**) &d_A, size);
	CUDA_ERROR(err, "Failed to allocate device vector d_A");
	err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	CUDA_ERROR(err, "Failed to copy vector h_A to d_A");

	//
	// H_BLOCK_SCAN
	//

	// TEST CASE

	int hbsTestSize = 10;
	int h_T_hbs_in[] = { 1, 2, 3, 4, 5, 1, 2, 3, 4, 5 };
	int h_T_hbs_exp[] = { 0, 1, 3, 6, 10, 0, 1, 3, 6, 10 };
	int testBlockSize = 5;
	int testBlocks = 1 + ((hbsTestSize - 1) / testBlockSize);
	int *h_T_hbs = (int*) malloc(sizeof(int) * hbsTestSize);

	sdkStartTimer(&timer);
	h_block_scan(hbsTestSize, testBlocks, testBlockSize, h_T_hbs_in, h_T_hbs);
	sdkStopTimer(&timer);

	h_msecs = sdkGetTimerValue(&timer);
	printf("[H_BLOCK_SCAN] for test case in %.5fmSecs\n", h_msecs);

	bool resultCorrectHBS = correct_results_full(hbsTestSize, h_T_hbs, h_T_hbs_exp);
	if (resultCorrectHBS)
		printf("Test passed\n");
	else
		exit(EXIT_FAILURE);

	free(h_T_hbs);

	// ACTUAL CALCULATION

	int *h_B_hbs = (int*) malloc(size);

	sdkStartTimer(&timer);
	h_block_scan(numElements, numSegments, segSize, h_A, h_B_hbs);
	sdkStopTimer(&timer);

	h_msecs = sdkGetTimerValue(&timer);
	printf("[H_BLOCK_SCAN] for %d elements in %.5fmSecs\n", numElements, h_msecs);

	//
	// D_BLOCK_SCAN (listing 2)
	//

	int *d_B = NULL;
	err = hipMalloc((void**) &d_B, size);
	CUDA_ERROR(err, "Failed to allocate device vector d_B");

	size_t sumsSize = numSegments * sizeof(int);
	int *d_SUMS = NULL;
	err = hipMalloc((void**) &d_SUMS, sumsSize);

	hipEventRecord(start, 0);
	d_block_scan<<<numSegments, blockSize>>>(numElements, d_A, d_B, d_SUMS);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipDeviceSynchronize();
	err = hipGetLastError();
	CUDA_ERROR(err, "Failed to launch d_block_scan kernel");

	err = hipEventElapsedTime(&d_msecs, start, stop);
	CUDA_ERROR(err, "Failed to get elapsed time");
	printf("[D_BLOCK_SCAN] Executed block scan in %d blocks of %d threads in = %.5fmSecs\n", numSegments, blockSize, d_msecs);

	// Verify result against result of h_block_scan
	int *h_B_dbs = (int*) malloc(size);
	err = hipMemcpy(h_B_dbs, d_B, size, hipMemcpyDeviceToHost);
	CUDA_ERROR(err, "Failed to copy vector d_B to h_B_dbs");
	if (correct_results_full(numElements, h_B_dbs, h_B_hbs))
		printf("Test passed\n");
	else
		exit(EXIT_FAILURE);
	free(h_B_dbs);

	err = hipFree(d_SUMS);
	CUDA_ERROR(err, "Failed to free device vector d_SUMS");
	err = hipFree(d_B);
	CUDA_ERROR(err, "Failed to free device vector d_B");

	//
	// H_FULL_SCAN
	//

	// TEST CASE (from paper)

	int hfsTestSize = 8;
	int h_T_hfs_in[] = { 3, 1, 7, 0, 4, 1, 6, 3 };
	int h_T_hfs_exp[] = { 0, 3, 4, 11, 11, 15, 16, 22 };
	int *h_T_hfs = (int*) malloc(sizeof(int) * hfsTestSize);

	sdkStartTimer(&timer);
	h_full_scan(hfsTestSize, h_T_hfs_in, h_T_hfs);
	sdkStopTimer(&timer);

	h_msecs = sdkGetTimerValue(&timer);
	printf("[H_FULL_SCAN] for test case in %.5fmSecs\n", h_msecs);

	bool resultCorrectHFS = correct_results_full(hfsTestSize, h_T_hfs, h_T_hfs_exp);
	if (resultCorrectHFS)
		printf("Test passed\n");
	else
		exit(EXIT_FAILURE);

	free(h_T_hfs);

	// ACTUAL CALCULATION

	int *h_B_hfs = (int*) malloc(size);

	sdkStartTimer(&timer);
	h_full_scan(numElements, h_A, h_B_hfs);
	sdkStopTimer(&timer);

	h_msecs = sdkGetTimerValue(&timer);
	printf("[H_FULL_SCAN] for %d elements in %.5fmSecs\n", numElements, h_msecs);

	//
	// D_FULL_SCAN
	//

	err = hipMalloc((void**) &d_B, size);
	CUDA_ERROR(err, "Failed to allocate device vector d_B");

	hipEventRecord(start, 0);
	d_full_scan(numElements, numSegments, segSize, d_A, d_B);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	err = hipEventElapsedTime(&d_msecs, start, stop);
	CUDA_ERROR(err, "Failed to get elapsed time");
	printf("[D_FULL_SCAN] Executed full scan in %d blocks of %d threads in = %.5fmSecs\n", numSegments, blockSize, d_msecs);

	// Verify result against result of h_full_scan
	int *h_B_dfs = (int*) malloc(size);
	err = hipMemcpy(h_B_dfs, d_B, size, hipMemcpyDeviceToHost);
	CUDA_ERROR(err, "Failed to copy vector d_B to h_B_dfs");
	if (correct_results_full(numElements, h_B_dfs, h_B_hfs))
		printf("Test passed\n");
	else
		exit(EXIT_FAILURE);
	free(h_B_dfs);

	err = hipFree(d_B);
	CUDA_ERROR(err, "Failed to free device vector d_B");

	//
	// TEARDOWN
	//

	// Free device input vector d_A
	err = hipFree(d_A);
	CUDA_ERROR(err, "Failed to free device vector d_A");

	// Free host memory
	free(h_B_hbs);
	free(h_B_hfs);
	free(h_A);

	// Clean up the Host timer
	sdkDeleteTimer(&timer);

	// Clean up the Device timer event objects
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Reset the device and exit
	err = hipDeviceReset();
	CUDA_ERROR(err, "Failed to reset the device");

	printf("Done\n");
	return EXIT_SUCCESS;
}
